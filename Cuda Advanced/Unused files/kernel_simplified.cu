﻿#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <iostream>

#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\load_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\preprocess_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\verify_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\convolution.cu>
//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\max_pooling.cu>
//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\activations.cu>


#define IMG_SIZE 32*32*3 // 32x32x3
#define NUM_IMAGES 10000 // 10000 images per batch
#define DATA_BATCHES 5   // Total number of data batches


void gpu_mem_info() {
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    std::cout << "\nGPU memory usage: used = " << used_db / 1024.0 / 1024.0 << "MB, free = " << free_db / 1024.0 / 1024.0 << "MB, total = " << total_db / 1024.0 / 1024.0 << "MB" << std::endl;
}


int main() {
    // Step 1. Load data
    unsigned char* d_images = nullptr;
    unsigned char* d_labels = nullptr;
    std::tie(d_images, d_labels) = load_data();
    if (d_images == nullptr || d_labels == nullptr) {
        std::cerr << "Failed to load data" << std::endl;
        return -1;
    }

    printf("Priting values just after load_data()\n");
 
	// Step 2. PREPROCESS DATA
    // Convert data to float and normalize
    float* d_images_float = nullptr;
    float* d_labels_float = nullptr;
    preprocessImage(d_images, &d_images_float, d_labels, &d_labels_float);

    gpu_mem_info();
    hipFree(d_images);
    hipFree(d_labels);

    //  Step3. CONVOLUTION
    int inputWidth = 32, inputHeight = 32, inputChannels = 3;

    ConvolutionLayer conv1(inputWidth, inputHeight, inputChannels, NUM_IMAGES);
	float* conv_pass = conv1.forward(d_images_float); // Forward pass

	int poolOutputWidth = conv1.getPoolOutputWidth();
	int poolOutputHeight = conv1.getPoolOutputHeight();
    int poolOutputChannels = conv1.getPoolOutputChannels();

	printf("\nPOOL 1 resutls - external");
	printf("\nOutput width: , Output height: , Output channels: %d %d %d\n", poolOutputWidth, poolOutputHeight, poolOutputChannels);

    return 0;
}

