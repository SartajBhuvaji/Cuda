#include <hip/hip_runtime.h>

#include <iostream>

// Kernel for SGD update
__global__ void sgdWeightsKernel(float* d_weights, float* d_gradients, float* d_velocity,
                                int size, float learningRate, float momentum) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_velocity[idx] = momentum * d_velocity[idx] - learningRate * d_gradients[idx];
        d_weights[idx] += d_velocity[idx];
    }
}

void sgdUpdateWeights(float* d_weights, float* d_gradients, float* d_velocity,
                     int size, float learningRate, float momentum) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    sgdWeightsKernel<<<numBlocks, blockSize>>>(d_weights, d_gradients, d_velocity,
                                              size, learningRate, momentum);
    hipDeviceSynchronize();
}

__global__ void sgdBiasesKernel(float* d_biases, float* d_grad_biases, int size, float learningRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_biases[idx] -= learningRate * d_grad_biases[idx];
    }
}

void sgdUpdateBiases(float* d_biases, float* d_grad_biases, int size, float learningRate) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    sgdBiasesKernel<<<numBlocks, blockSize>>>(d_biases, d_grad_biases, size, learningRate);
    hipDeviceSynchronize();
} 