#include <hip/hip_runtime.h>

#include <iostream>

// Kernel for SGD update
__global__ void sgdWeightsKernel(float* d_weights, float* d_gradients, int size, float learningRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_weights[idx] -= learningRate * d_gradients[idx];
    }
}

void sgdUpdateWeights(float* d_weights, float* d_grad_weights, int size, float learningRate) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    sgdWeightsKernel<<<numBlocks, blockSize>>>(d_weights, d_grad_weights, size, learningRate);
    hipDeviceSynchronize();
}

__global__ void sgdBiasesKernel(float* d_biases, float* d_grad_biases, int size, float learningRate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        d_biases[idx] -= learningRate * d_grad_biases[idx];
    }
}

void sgdUpdateBiases(float* d_biases, float* d_grad_biases, int size, float learningRate) {
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    sgdBiasesKernel<<<numBlocks, blockSize>>>(d_biases, d_grad_biases, size, learningRate);
    hipDeviceSynchronize();
} 