﻿#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>
#include <cstdlib>

// Kernel for dense layer forward pass
__global__ void denseForwardKernel(float* input, float* weights, float* biases, float* output, int inputSize, int outputSize, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batchSize && col < outputSize) {
        float sum = 0.0f;
        for (int i = 0; i < inputSize; ++i) {
            sum += input[row * inputSize + i] * weights[i * outputSize + col];
        }
        output[row * outputSize + col] = sum + biases[col];
    }
}

class DenseLayer {
private:
    int inputSize, outputSize, batchSize;
    float* d_weights;  // Device memory for weights
    float* d_biases;   // Device memory for biases
    float* d_output;   // Device memory for output

public:
    DenseLayer(int inSize, int outSize, int batchSize)
        : inputSize(inSize), outputSize(outSize), batchSize(batchSize) {
        // Allocate memory for weights and biases
        hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_biases, outputSize * sizeof(float));
        hipMalloc(&d_output, outputSize * batchSize * sizeof(float));

        // Initialize weights and biases
        initializeParameters();
    }

    ~DenseLayer() {
        hipFree(d_weights);
        hipFree(d_biases);
        hipFree(d_output);
    }

    void initializeParameters() {
        // Initialize weights using Xavier initialization
        float scale = sqrt(2.0f / (inputSize + outputSize));
        float* h_weights = new float[inputSize * outputSize];
        float* h_biases = new float[outputSize];

        for (int i = 0; i < inputSize * outputSize; ++i) {
            h_weights[i] = scale * ((float)rand() / RAND_MAX * 2.0f - 1.0f);
        }
        for (int i = 0; i < outputSize; ++i) {
            h_biases[i] = scale * ((float)rand() / RAND_MAX * 2.0f - 1.0f);
        }

        hipMemcpy(d_weights, h_weights, inputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_biases, h_biases, outputSize * sizeof(float), hipMemcpyHostToDevice);

        delete[] h_weights;
        delete[] h_biases;
    }

    float* forward(float* d_input) {
        // Perform matrix multiplication and add biases
        dim3 blockDim(16, 16);
        dim3 gridDim((outputSize + blockDim.x - 1) / blockDim.x, (batchSize + blockDim.y - 1) / blockDim.y);

        // Launch a kernel to perform the forward pass
        denseForwardKernel<<<gridDim, blockDim>>>(d_input, d_weights, d_biases, d_output, inputSize, outputSize, batchSize);

        hipDeviceSynchronize();
        return d_output;
    }

    // Placeholder for backpropagation function
    void backpropagate(const float* gradients, float learningRate) {
        // TODO: Implement backpropagation
        // This will update weights and biases based on the gradients
    }

    int getOutputSize() const { return outputSize; }
    int getBatchSize() const { return batchSize; }
    float* getWeights() const { return d_weights; }
    float* getBias() const { return d_biases; }
};