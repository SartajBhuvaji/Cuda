#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <cmath>
#include <vector>
//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\activations.cu>

// Custom CUDA kernel for weight initialization
__global__ void initializeWeightsKernel(float* weights, float* bias, int inputSize, int outputSize, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < inputSize * outputSize) {
        // Simple random number generation using XORShift
        unsigned int state = seed + idx;
        state ^= (state << 13);
        state ^= (state >> 17);
        state ^= (state << 5);

        // Generate a random number between -1 and 1
        float random = (float)state / 4294967295.0f * 2.0f - 1.0f;

        // He initialization
        float std_dev = sqrtf(2.0f / inputSize);
        weights[idx] = random * std_dev;
    }

    if (idx < outputSize) {
        // Initialize bias
        unsigned int state = seed + idx + inputSize * outputSize;
        state ^= (state << 13);
        state ^= (state >> 17);
        state ^= (state << 5);

        float random = (float)state / 4294967295.0f * 2.0f - 1.0f;
        float std_dev = sqrtf(2.0f / inputSize);
        bias[idx] = random * std_dev;
    }
}


__global__ void forwardKernel(const float* input, const float* weights, const float* bias,
    float* output, int inputSize, int outputSize, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.y;
    if (idx < outputSize && batch < batchSize) {
        float sum = 0.0f;
        for (int i = 0; i < inputSize; ++i) {
            sum += input[batch * inputSize + i] * weights[i * outputSize + idx];
        }
        output[batch * outputSize + idx] = sum + bias[idx];
    }
}

class DenseLayer {
private:
    int inputSize;
    int outputSize;
    int batchSize;
    float* d_weights;
    float* d_bias;
    float* d_output;
    float* d_input;
    const char* activationType;

    void initializeWeightsAndBiases() {
        int blockSize = 256;
        int numBlocks = (inputSize * outputSize + blockSize - 1) / blockSize;
        initializeWeightsKernel << <numBlocks, blockSize >> > (d_weights, d_bias, inputSize, outputSize, 1234);
        hipDeviceSynchronize();
    }

public:
    DenseLayer(int inSize, int outSize, int bSize, const char* actType)
        : inputSize(inSize), outputSize(outSize), batchSize(bSize), activationType(actType) {
        hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_bias, outputSize * sizeof(float));
        hipMalloc(&d_output, outputSize * batchSize * sizeof(float));
        hipMalloc(&d_input, inputSize * batchSize * sizeof(float));
        initializeWeightsAndBiases();
    }

    ~DenseLayer() {
        hipFree(d_weights);
        hipFree(d_bias);
        hipFree(d_output);
        hipFree(d_input);
    }

    float* forward(const float* input) {
        float h_input[10];
        hipMemcpy(h_input, input, 10 * sizeof(float), hipMemcpyDeviceToHost);
        printf("Dense layer input (first 10 values):\n");
        for (int i = 0; i < 10; ++i) {
            printf("%f ", h_input[i]);
        }
        printf("\n");

        hipMemcpy(d_input, input, inputSize * batchSize * sizeof(float), hipMemcpyDeviceToDevice);

        dim3 blockDim(256);
        dim3 gridDim((outputSize + blockDim.x - 1) / blockDim.x, batchSize);

        forwardKernel << <gridDim, blockDim >> > (input, d_weights, d_bias, d_output, inputSize, outputSize, batchSize);
        hipDeviceSynchronize();

        // Apply activation function
        float* d_activated_output;
        hipMalloc(&d_activated_output, outputSize * batchSize * sizeof(float));

        applyActivation(d_output, d_activated_output, outputSize * batchSize, activationType);
        

        // Debugging: Print some values from d_output and d_activated_output for the first batch
        float h_output[10];
        float h_activated_output[10];
        hipMemcpy(h_output, d_output, 10 * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_activated_output, d_activated_output, 10 * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "Before activation (first batch):" << std::endl;
        for (int i = 0; i < 10; ++i) {
            std::cout << h_output[i] << " ";
        }
        std::cout << std::endl;

        std::cout << "After activation (first batch):" << std::endl;
        for (int i = 0; i < 10; ++i) {
            std::cout << h_activated_output[i] << " ";
        }
        std::cout << std::endl;

        hipFree(d_output);
        d_output = d_activated_output;
        return d_output;
    }

    // Placeholder for backpropagation function
    void backpropagate(const float* gradients, float learningRate) {
        // TODO: Implement backpropagation
        // This will update weights and biases based on the gradients
    }

    int getOutputSize() const { return outputSize; }
    int getBatchSize() const { return batchSize; }
    float* getWeights() const { return d_weights; }
    float* getBias() const { return d_bias; }
};

// Example usage remains the same
float* runNeuralNetwork(float* input, int inputSize, int hiddenSize, int numLayers, int outputSize, int batchSize) {
    std::vector<DenseLayer*> layers;
    // Create layers
    layers.push_back(new DenseLayer(inputSize, hiddenSize, batchSize, "relu"));
    for (int i = 1; i < numLayers - 1; ++i) {
        printf("Creating hidden layer %d\n", i);
        layers.push_back(new DenseLayer(hiddenSize, hiddenSize, batchSize, "relu"));
    }
    layers.push_back(new DenseLayer(hiddenSize, outputSize, batchSize, "softmax"));

    // Forward pass
    printf("numLayers: %d\n", numLayers);
    float* layerInput = input;
    for (int i = 0; i < numLayers; ++i) {
        std::cout << "Layer " << i << " output:" << std::endl;
        layerInput = layers[i]->forward(layerInput);
    }

    // Print the output of the last layer for the first batch
    printf("Output of the last layer (first batch):\n");
    float* output = new float[layers[numLayers - 1]->getOutputSize() * batchSize];
    hipMemcpy(output, layerInput, layers[numLayers - 1]->getOutputSize() * batchSize * sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < layers[numLayers - 1]->getOutputSize(); ++i) {
        printf("%f ", output[i]);
    }
    printf("\n");

	return output;

    // Clean up
    //for (auto& layer : layers) {
    //    delete layer;
    //}
    //delete[] output;
}