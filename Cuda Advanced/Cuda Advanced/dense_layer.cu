﻿#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>
#include <cstdlib>

// Kernel for dense layer forward pass
__global__ void denseForwardKernel(float* input, float* weights, float* biases, float* output, int inputSize, int outputSize, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batchSize && col < outputSize) {
        float sum = 0.0f;
        for (int i = 0; i < inputSize; ++i) {
            sum += input[row * inputSize + i] * weights[i * outputSize + col];
        }
        output[row * outputSize + col] = sum + biases[col];
    }
}

// Kernel for dense layer backward pass
__global__ void denseBackwardKernel(float* input, float* gradients, float* grad_weights, float* grad_biases, 
                                   int inputSize, int outputSize, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < inputSize && col < outputSize) {
        // Calculate gradient for weights
        float grad_w = 0.0f;
        for (int b = 0; b < batchSize; ++b) {
            grad_w += input[b * inputSize + row] * gradients[b * outputSize + col];
        }
        grad_weights[row * outputSize + col] = grad_w;

        // Calculate gradient for biases (only one thread per column)
        if (row == 0) {
            float grad_b = 0.0f;
            for (int b = 0; b < batchSize; ++b) {
                grad_b += gradients[b * outputSize + col];
            }
            grad_biases[col] = grad_b;
        }
    }
}

// Add gradient propagation kernel
__global__ void gradientPropagationKernel(float* input_gradients, float* output_gradients, 
                                         float* weights, int inputSize, int outputSize, int batchSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int batch = blockIdx.y;
    
    if (idx < inputSize && batch < batchSize) {
        float sum = 0.0f;
        for (int j = 0; j < outputSize; ++j) {
            sum += output_gradients[batch * outputSize + j] * weights[idx * outputSize + j];
        }
        input_gradients[batch * inputSize + idx] = sum;
    }
}

class DenseLayer {
private:
    int inputSize, outputSize, batchSize;
    float* d_weights;  // Device memory for weights
    float* d_biases;   // Device memory for biases
    float* d_output;   // Device memory for output
    float* d_grad_weights;  // Device memory for weight gradients
    float* d_grad_biases;   // Device memory for bias gradients
    float* d_velocity_weights;  // Add momentum
    float* d_velocity_biases;
    const float momentum = 0.9f;
    float* d_input_gradients;  // Add this for storing input gradients

public:
    DenseLayer(int inSize, int outSize, int batchSize)
        : inputSize(inSize), outputSize(outSize), batchSize(batchSize) {
        // Allocate memory for weights and biases
        hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_biases, outputSize * sizeof(float));
        hipMalloc(&d_output, outputSize * batchSize * sizeof(float));
        hipMalloc(&d_grad_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_grad_biases, outputSize * sizeof(float));
        hipMalloc(&d_velocity_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_velocity_biases, outputSize * sizeof(float));
        hipMemset(d_velocity_weights, 0, inputSize * outputSize * sizeof(float));
        hipMemset(d_velocity_biases, 0, outputSize * sizeof(float));
        hipMalloc(&d_input_gradients, inputSize * batchSize * sizeof(float));

        // Initialize weights and biases
        initializeParameters();
    }

    ~DenseLayer() {
        hipFree(d_weights);
        hipFree(d_biases);
        hipFree(d_output);
        hipFree(d_grad_weights);
        hipFree(d_grad_biases);
        hipFree(d_velocity_weights);
        hipFree(d_velocity_biases);
        hipFree(d_input_gradients);
    }

    void initializeParameters() {
        // Initialize weights using He initialization
        float scale = sqrt(2.0f / inputSize);  // He initialization
        float* h_weights = new float[inputSize * outputSize];
        float* h_biases = new float[outputSize];

        for (int i = 0; i < inputSize * outputSize; ++i) {
            h_weights[i] = scale * ((float)rand() / RAND_MAX * 2.0f - 1.0f);
        }
        
        // Initialize biases to small values
        for (int i = 0; i < outputSize; ++i) {
            h_biases[i] = 0.01f * ((float)rand() / RAND_MAX * 2.0f - 1.0f);
        }

        hipMemcpy(d_weights, h_weights, inputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_biases, h_biases, outputSize * sizeof(float), hipMemcpyHostToDevice);

        delete[] h_weights;
        delete[] h_biases;
    }

    float* forward(float* d_input) {
        // Perform matrix multiplication and add biases
        dim3 blockDim(16, 16);
        dim3 gridDim((outputSize + blockDim.x - 1) / blockDim.x, (batchSize + blockDim.y - 1) / blockDim.y);

        // Launch a kernel to perform the forward pass
        denseForwardKernel<<<gridDim, blockDim>>>(d_input, d_weights, d_biases, d_output, inputSize, outputSize, batchSize);

        hipDeviceSynchronize();
        return d_output;
    }


    int getOutputSize() const { return outputSize; }
    int getBatchSize() const { return batchSize; }
    float* getWeights() const { return d_weights; }
    float* getBiases() const { return d_biases; }
    float* getGradWeights() const { return d_grad_weights; }
    float* getGradBiases() const { return d_grad_biases; }

    void backward(float* d_input, float* d_gradients) {
        // Reset gradients
        hipMemset(d_grad_weights, 0, inputSize * outputSize * sizeof(float));
        hipMemset(d_grad_biases, 0, outputSize * sizeof(float));
        hipMemset(d_input_gradients, 0, inputSize * batchSize * sizeof(float));

        // Calculate gradients for weights and biases
        dim3 blockDim(16, 16);
        dim3 gridDim(
            (outputSize + blockDim.x - 1) / blockDim.x,
            (inputSize + blockDim.y - 1) / blockDim.y
        );

        denseBackwardKernel<<<gridDim, blockDim>>>(
            d_input, d_gradients, d_grad_weights, d_grad_biases,
            inputSize, outputSize, batchSize
        );

        // Propagate gradients to previous layer
        dim3 propBlockDim(256);
        dim3 propGridDim(
            (inputSize + propBlockDim.x - 1) / propBlockDim.x,
            batchSize
        );

        gradientPropagationKernel<<<propGridDim, propBlockDim>>>(
            d_input_gradients, d_gradients, d_weights,
            inputSize, outputSize, batchSize
        );

        hipDeviceSynchronize();
    }

    float* getInputGradients() const { return d_input_gradients; }
    float* getVelocityWeights() const { return d_velocity_weights; }
    float* getVelocityBiases() const { return d_velocity_biases; }
};