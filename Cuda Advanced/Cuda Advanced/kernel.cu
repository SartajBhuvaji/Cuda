﻿#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <iostream>

#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\load_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\preprocess_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\verify_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\convolution.cu>
//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\max_pooling.cu>
//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\activations.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\dense_layer.cu>


#define IMG_SIZE 32*32*3 // 32x32x3
#define NUM_IMAGES 10000 // 10000 images per batch
#define DATA_BATCHES 5   // Total number of data batches


void gpu_mem_info() {

    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    std::cout << "\nGPU memory usage: used = " << used_db / 1024.0 / 1024.0 << "MB, free = " << free_db / 1024.0 / 1024.0 << "MB, total = " << total_db / 1024.0 / 1024.0 << "MB" << std::endl;
}



int main() {
    // Step 1. Load data
    unsigned char* d_images = nullptr;
    unsigned char* d_labels = nullptr;
    std::tie(d_images, d_labels) = load_data();
    if (d_images == nullptr || d_labels == nullptr) {
        std::cerr << "Failed to load data" << std::endl;
        return -1;
    }

    printf("Priting values just after load_data()\n");
    unsigned char* h_images = (unsigned char*)malloc(IMG_SIZE * NUM_IMAGES * DATA_BATCHES);
    hipMemcpy(h_images, d_images, IMG_SIZE * NUM_IMAGES * DATA_BATCHES, hipMemcpyDeviceToHost);
    for (int i = 0; i < 100; i++) {
        printf("%d ", (int)h_images[i]);
    }
    printf("\n");

    // Convert data to float and normalize
    float* d_images_float = nullptr;
    float* d_labels_float = nullptr;
    preprocessImage(d_images, &d_images_float, d_labels, &d_labels_float);

    gpu_mem_info();

    hipFree(d_images);
    hipFree(d_labels);



    // copy from device to host
    float* h_labels_float = (float*)malloc(NUM_IMAGES * DATA_BATCHES * sizeof(float));
    //float* h_images_float = (float*)malloc(IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float));
    float* h_images_float = (float*)malloc(IMG_SIZE * NUM_IMAGES * DATA_BATCHES * sizeof(float));

    hipMemcpy(h_labels_float, d_labels_float, NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_images_float, d_images_float, IMG_SIZE * NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);

    //hipMemcpy(h_images_float, d_images_float, IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);
    
    // print the first 10 labels


    // Define batch size
    const int BATCH_SIZE = 100; // Adjust this value based on your GPU memory
    const int NUM_BATCHES = NUM_IMAGES * DATA_BATCHES / BATCH_SIZE;

    // Create a convolution layer
    int inputWidth = 32, inputHeight = 32, inputChannels = 3;
    ConvolutionLayer conv1(inputWidth, inputHeight, inputChannels, BATCH_SIZE);


    // Create dense layers
    int convOutputSize = conv1.getPoolOutputWidth() * conv1.getPoolOutputHeight() * conv1.getPoolOutputChannels();
    int hiddenSize = 64;
    int numLayers = 3; // 1 input layer, 1 hidden layer, 1 output layer
    int outputSize = 10; // Assuming 10 classes for classification

    std::vector<DenseLayer*> denseLayers;
    denseLayers.push_back(new DenseLayer(convOutputSize, hiddenSize, BATCH_SIZE, "softmax"));
    //denseLayers.push_back(new DenseLayer(hiddenSize, hiddenSize, BATCH_SIZE, "relu"));
    //denseLayers.push_back(new DenseLayer(hiddenSize, outputSize, BATCH_SIZE, "softmax"));

    //NUM_BATCHES
    for (int batch = 0; batch < 2; ++batch) {
        // Calculate the offset for the current batch
        size_t batchOffset = batch * BATCH_SIZE * IMG_SIZE;

        // Pointer to the current batch of images
        float* d_batch_images = d_images_float + batchOffset;

        // Perform forward pass for the current batch through convolution layer
        float* conv_output = conv1.forward(d_batch_images);

        printf("\nBatch %d - CONV 1 results:", batch);
        printf("\nOutput width: %d, Output height: %d, Output channels: %d\n",
            conv1.getPoolOutputWidth(), conv1.getPoolOutputHeight(), conv1.getPoolOutputChannels());

        // Forward pass through dense layers
        float* denseInput = conv_output;
        for (int i = 0; i < denseLayers.size(); ++i) {
            denseInput = denseLayers[i]->forward(denseInput);
            printf("Dense Layer %d output (first few values of first batch):\n", i);
            float h_output[10];
            hipMemcpy(h_output, denseInput, 10 * sizeof(float), hipMemcpyDeviceToHost);
            for (int j = 0; j < 10; ++j) {
                printf("%f ", h_output[j]);
            }
            printf("\n");
        }

        // The final output is now in denseInput
        // TODO: Implement loss calculation and backpropagation

        // Free the memory allocated for conv_output if it's no longer needed
        hipFree(conv_output);
    }



 //   // print the first image
 //   int counter = 0;
 //   printf("First image before convolution\n");
 //   for (int i = 0; i < 1; i++) {
 //       for (int j = 0; j < IMG_SIZE; j++) {
 //           std::cout << h_images_float[j + i * IMG_SIZE] << " ";
 //           counter++;
 //       }
 //       std::cout << std::endl;
 //   }
 //   printf("Total number of pixels: %d\n", counter);

 //   //  CONVOLUTION
 //   int inputWidth = 32, inputHeight = 32, inputChannels = 3;

 //   ConvolutionLayer conv1(inputWidth, inputHeight, inputChannels, NUM_IMAGES);
 //   // Perform forward pass
 //   float* conv_pass = conv1.forward(d_images_float);

 //   // Allocate host memory for the output
 //   /*int conv1outputWidth = conv1.getOutputWidth();
 //   int conv1outputHeight = conv1.getOutputHeight();
 //   int conv1outputChannels = conv1.getOutputChannels();*/


	//int poolOutputWidth = conv1.getPoolOutputWidth();
	//int poolOutputHeight = conv1.getPoolOutputHeight();
 //   int poolOutputChannels = conv1.getPoolOutputChannels();

	//printf("\nPOOL 1 resutls - external");
	//printf("\nOutput width: , Output height: , Output channels: %d %d %d\n", poolOutputWidth, poolOutputHeight, poolOutputChannels);

 //   //DENSE LAYER
	//runNeuralNetwork(conv_pass, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES, 64, 5, 10);








    //float* conv1h_output = (float*)malloc(conv1outputWidth * conv1outputHeight * conv1outputChannels * NUM_IMAGES * sizeof(float));
    /*float* conv1h_conv_filter = (float*)malloc(FILTER_SIZE * FILTER_SIZE * inputChannels * conv1outputChannels * sizeof(float));*/
    //printf("Output width: , Output height: , Output channels: %d %d %d\n", conv1outputWidth, conv1outputHeight, conv1outputChannels);

    // Copy the result back to host
    // hipMemcpy(conv1h_output, conv1d_output_conv, conv1outputWidth * conv1outputHeight * conv1outputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first image after convolution
    //counter = 0;
    //printf("First image after convolution\n");
    //for (int c = 0; c < conv1outputChannels; ++c) {
    //    for (int i = 0; i < conv1outputHeight; ++i) {
    //        for (int j = 0; j < conv1outputWidth; ++j) {
    //            //std::cout << conv1h_output[(c * conv1outputHeight * conv1outputWidth) + (i * conv1outputWidth) + j] << " ";
    //            counter++;
    //        }
    //        // std::cout << std::endl;
    //    }
    //    //std::cout << "Channel " << outputChannels << " complete" << std::endl;
    //}
    //printf("Total number of pixels after conv1: %d\n", counter);


    //MAX POOLING
    //MaxPoolingLayer pool1(conv1.getOutputWidth(), conv1.getOutputHeight(), conv1.getOutputChannels(), NUM_IMAGES);
    //float* d_pool_output = pool1.forward(conv1d_output_conv);

    //int poolOutputWidth = pool1.getOutputWidth();
    //int poolOutputHeight = pool1.getOutputHeight();
    //int poolOutputChannels = pool1.getOutputChannels();
    //float* h_pool_output = (float*)malloc(poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));

    //printf("\nPOOL 1 resutls");
    //printf("\nOutput width: , Output height: , Output channels: %d %d %d\n", poolOutputWidth, poolOutputHeight, poolOutputChannels);

    //hipMemcpy(h_pool_output, d_pool_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

	// ACTIVATION
	/*float* d_activated_output;
	hipMalloc(&d_activated_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));
	applyActivation(d_pool_output, d_activated_output, poolOutputWidth* poolOutputHeight* poolOutputChannels* NUM_IMAGES, "relu");
     */

	//// Copy the result back to host
	//float* h_activated_output = (float*)malloc(poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));
	//printf("\nACTIVATION results");
	//hipMemcpy(h_activated_output, d_activated_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

	//// Print the first image after convolution
	//counter = 0;
	//printf("\n\nFirst image after activation\n");
 //   for (int c = 0; c < poolOutputChannels; ++c) {
 //       for (int i = 0; i < poolOutputHeight; ++i) {
 //           for (int j = 0; j < poolOutputWidth; ++j) {
 //               std::cout << h_activated_output[(c * poolOutputHeight * poolOutputWidth) + (i * poolOutputWidth) + j] << " ";
 //               counter++;
 //           }
 //           std::cout << std::endl;
 //       }
 //   }







    return 0;
}


/*

float* d_images_gray_norm;
float* d_labels_float;
hipMalloc(&d_images_gray_norm, IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float));
hipMalloc(&d_labels_float, NUM_IMAGES * DATA_BATCHES * sizeof(float));

preprocessImages(d_images, d_images_gray_norm, d_labels, d_labels_float);
verifyGrayscaleConversion(d_images_gray_norm, d_labels_float);

// Free memory on gpu
/*hipFree(d_images);
hipFree(d_labels);


float* d_output;
hipMalloc(&d_output, (IMG_WIDTH - 2) * (IMG_HEIGHT - 2) * NUM_IMAGES * DATA_BATCHES * sizeof(float));
perform_convolution(d_images_gray_norm, d_labels_float, NUM_IMAGES * DATA_BATCHES);

// Verify grayscale conversion, normalization, and convolution
verify_grayscale_normalization(d_images_gray_norm, d_labels_float, NUM_IMAGES * DATA_BATCHES);

// Clean up
hipFree(d_output);

*/