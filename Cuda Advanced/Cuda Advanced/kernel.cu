﻿#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <tuple>

#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\load_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\preprocess_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\verify_images.cu>

#define IMG_SIZE 32*32*3 // 32x32x3
#define NUM_IMAGES 10000 // 10000 images per batch
#define DATA_BATCHES 5      // Total number of data batches


int main() {
    // Step1. Load data
    unsigned char* d_images = nullptr;
    unsigned char* d_labels = nullptr;
    std::tie(d_images, d_labels) = load_data();

    if (d_images == nullptr || d_labels == nullptr) {
        std::cerr << "Failed to load data" << std::endl;
        return 1;
    }

    // Step2. Pre-process data
    //unsigned char* d_gray = preprocess_image(d_images);

    // Verify GPU batch load
    verify_GPU_batch_load(d_images, d_labels);
	
    printf("Batch load verified\n");

    // Clean up
    hipFree(d_images);
    hipFree(d_labels);

    return 0;
}
