﻿#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <iostream>


#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\load_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\preprocess_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\verify_images.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\convolution.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\max_pooling.cu>
#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\activations.cu>

//#include<C:\\Users\\sbhuv\\Desktop\\Cuda\\Cuda\\Cuda Advanced\\Cuda Advanced\\dense_layers.cu>



#define IMG_SIZE 32*32*3 // 32x32x3
#define NUM_IMAGES 10000 // 10000 images per batch
#define DATA_BATCHES 5   // Total number of data batches


void gpu_mem_info() {

    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte, &total_byte);
    double free_db = (double)free_byte;
    double total_db = (double)total_byte;
    double used_db = total_db - free_db;
    std::cout << "\nGPU memory usage: used = " << used_db / 1024.0 / 1024.0 << "MB, free = " << free_db / 1024.0 / 1024.0 << "MB, total = " << total_db / 1024.0 / 1024.0 << "MB" << std::endl;
}


void convertAndDisplayImage(float* h_images_float, float* h_labels_float) {
    cv::Mat img(32, 32, CV_8UC3);
    for (int y = 0; y < 32; y++) {
        for (int x = 0; x < 32; x++) {
            for (int c = 0; c < 3; c++) {
                img.at<cv::Vec3b>(y, x)[c] = static_cast<char>(static_cast<int>(h_images_float[y * 32 + x + c * 1024] * 225.0f));
            }
        }
    }
    // print RGB value of the first pixel
    printf("RGB: %d %d %d\n", img.at<cv::Vec3b>(0, 0)[0], img.at<cv::Vec3b>(0, 0)[1], img.at<cv::Vec3b>(0, 0)[2]);
    cv::resize(img, img, cv::Size(250, 250));
    cv::imshow("Image", img);
    printf("Label: %d\n", h_labels_float[0]);
    cv::waitKey(5000);
}


void convertAndDisplayImage_old(float* h_images_float, int imageIndex, int width, int height) {
    cv::Mat grayscaleImage(height, width, CV_32F);

    // Copy the image data into the cv::Mat object
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            grayscaleImage.at<float>(i, j) = h_images_float[imageIndex * width * height + i * width + j];
        }
    }

    // Normalize the image to 0-255 range
    cv::Mat normalizedImage;
    cv::normalize(grayscaleImage, normalizedImage, 0, 255, cv::NORM_MINMAX, CV_8U);

    cv::Mat resizedImage;
    cv::resize(normalizedImage, resizedImage, cv::Size(720, 720), 0, 0, cv::INTER_NEAREST);
    cv::imshow("Grayscale Image", resizedImage);
    cv::waitKey(0);
    cv::destroyAllWindows();
}

int main() {
    // Step 1. Load data
    unsigned char* d_images = nullptr;
    unsigned char* d_labels = nullptr;
    std::tie(d_images, d_labels) = load_data();
    if (d_images == nullptr || d_labels == nullptr) {
        std::cerr << "Failed to load data" << std::endl;
        return -1;
    }

    printf("Priting values just after load_data()\n");
    unsigned char* h_images = (unsigned char*)malloc(IMG_SIZE * NUM_IMAGES * DATA_BATCHES);
    hipMemcpy(h_images, d_images, IMG_SIZE * NUM_IMAGES * DATA_BATCHES, hipMemcpyDeviceToHost);
    for (int i = 0; i < 100; i++) {
        printf("%d ", (int)h_images[i]);
    }
    printf("\n");

    // Convert data to float and normalize
    float* d_images_float = nullptr;
    float* d_labels_float = nullptr;
    preprocessImage(d_images, &d_images_float, d_labels, &d_labels_float);

    gpu_mem_info();

    hipFree(d_images);
    hipFree(d_labels);

    // copy from device to host
    float* h_labels_float = (float*)malloc(NUM_IMAGES * DATA_BATCHES * sizeof(float));
    //float* h_images_float = (float*)malloc(IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float));
    float* h_images_float = (float*)malloc(IMG_SIZE * NUM_IMAGES * DATA_BATCHES * sizeof(float));

    hipMemcpy(h_labels_float, d_labels_float, NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_images_float, d_images_float, IMG_SIZE * NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);

    //hipMemcpy(h_images_float, d_images_float, IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float), hipMemcpyDeviceToHost);

    // print the first 10 labels
    for (int i = 0; i < 10; i++) {
        std::cout << h_labels_float[i] << std::endl;
    }

    // print the first image
    int counter = 0;
    printf("First image before convolution\n");
    for (int i = 0; i < 1; i++) {
        for (int j = 0; j < IMG_SIZE; j++) {
            std::cout << h_images_float[j + i * IMG_SIZE] << " ";
            counter++;
        }
        std::cout << std::endl;
    }
    printf("Total number of pixels: %d\n", counter);

    //  CONVOLUTION
    int inputWidth = 32, inputHeight = 32, inputChannels = 3;

    ConvolutionLayer conv1(inputWidth, inputHeight, inputChannels, NUM_IMAGES);
    // Perform forward pass
    float* conv1d_output_conv = conv1.forward(d_images_float);

    // Allocate host memory for the output
     int conv1outputWidth = conv1.getOutputWidth();
    int conv1outputHeight = conv1.getOutputHeight();
    int conv1outputChannels = conv1.getOutputChannels();
    //float* conv1h_output = (float*)malloc(conv1outputWidth * conv1outputHeight * conv1outputChannels * NUM_IMAGES * sizeof(float));
    /*float* conv1h_conv_filter = (float*)malloc(FILTER_SIZE * FILTER_SIZE * inputChannels * conv1outputChannels * sizeof(float));*/
    //printf("Output width: , Output height: , Output channels: %d %d %d\n", conv1outputWidth, conv1outputHeight, conv1outputChannels);

    // Copy the result back to host
    // hipMemcpy(conv1h_output, conv1d_output_conv, conv1outputWidth * conv1outputHeight * conv1outputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

    // Print the first image after convolution
    counter = 0;
    printf("First image after convolution\n");
    for (int c = 0; c < conv1outputChannels; ++c) {
        for (int i = 0; i < conv1outputHeight; ++i) {
            for (int j = 0; j < conv1outputWidth; ++j) {
                //std::cout << conv1h_output[(c * conv1outputHeight * conv1outputWidth) + (i * conv1outputWidth) + j] << " ";
                counter++;
            }
            // std::cout << std::endl;
        }
        //std::cout << "Channel " << outputChannels << " complete" << std::endl;
    }
    printf("Total number of pixels after conv1: %d\n", counter);


    //MAX POOLING
    MaxPoolingLayer pool1(conv1.getOutputWidth(), conv1.getOutputHeight(), conv1.getOutputChannels(), NUM_IMAGES);
    float* d_pool_output = pool1.forward(conv1d_output_conv);

    int poolOutputWidth = pool1.getOutputWidth();
    int poolOutputHeight = pool1.getOutputHeight();
    int poolOutputChannels = pool1.getOutputChannels();
    float* h_pool_output = (float*)malloc(poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));

    printf("\nPOOL 1 resutls");
    printf("\nOutput width: , Output height: , Output channels: %d %d %d\n", poolOutputWidth, poolOutputHeight, poolOutputChannels);

    //hipMemcpy(h_pool_output, d_pool_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

	// ACTIVATION
	float* d_activated_output;
	hipMalloc(&d_activated_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));
	applyActivation(d_pool_output, d_activated_output, poolOutputWidth* poolOutputHeight* poolOutputChannels* NUM_IMAGES, "relu");


	//// Copy the result back to host
	//float* h_activated_output = (float*)malloc(poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float));
	//printf("\nACTIVATION results");
	//hipMemcpy(h_activated_output, d_activated_output, poolOutputWidth * poolOutputHeight * poolOutputChannels * NUM_IMAGES * sizeof(float), hipMemcpyDeviceToHost);

	//// Print the first image after convolution
	//counter = 0;
	//printf("\n\nFirst image after activation\n");
 //   for (int c = 0; c < poolOutputChannels; ++c) {
 //       for (int i = 0; i < poolOutputHeight; ++i) {
 //           for (int j = 0; j < poolOutputWidth; ++j) {
 //               std::cout << h_activated_output[(c * poolOutputHeight * poolOutputWidth) + (i * poolOutputWidth) + j] << " ";
 //               counter++;
 //           }
 //           std::cout << std::endl;
 //       }
 //   }






    return 0;
}


/*

float* d_images_gray_norm;
float* d_labels_float;
hipMalloc(&d_images_gray_norm, IMG_SIZE / 3 * NUM_IMAGES * DATA_BATCHES * sizeof(float));
hipMalloc(&d_labels_float, NUM_IMAGES * DATA_BATCHES * sizeof(float));

preprocessImages(d_images, d_images_gray_norm, d_labels, d_labels_float);
verifyGrayscaleConversion(d_images_gray_norm, d_labels_float);

// Free memory on gpu
/*hipFree(d_images);
hipFree(d_labels);


float* d_output;
hipMalloc(&d_output, (IMG_WIDTH - 2) * (IMG_HEIGHT - 2) * NUM_IMAGES * DATA_BATCHES * sizeof(float));
perform_convolution(d_images_gray_norm, d_labels_float, NUM_IMAGES * DATA_BATCHES);

// Verify grayscale conversion, normalization, and convolution
verify_grayscale_normalization(d_images_gray_norm, d_labels_float, NUM_IMAGES * DATA_BATCHES);

// Clean up
hipFree(d_output);

*/