#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <>
#include <opencv2/opencv.hpp>
#include <iostream>

#define IMG_SIZE 32*32*3 // 32x32x3
#define NUM_IMAGES 10000 // 10000 images per batch
#define DATA_BATCHES 5


__global__ void rgbToGrayscale(unsigned char* d_rgb, unsigned char* d_gray, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = 3 * grayOffset;

        unsigned char r = d_rgb[rgbOffset];
        unsigned char g = d_rgb[rgbOffset + 1];
        unsigned char b = d_rgb[rgbOffset + 2];

        // Convert to grayscale using luminosity method
        d_gray[grayOffset] = static_cast<unsigned char>(0.21f * r + 0.71f * g + 0.07f * b);
    }
}

unsigned char* preprocess_image(unsigned char*  d_images){ 
    int width = 32;
    int height = 32;

    // Allocate memory for grayscale image
    unsigned char* d_gray;
    hipMalloc(&d_gray, width* height * sizeof(unsigned char));

    // Define block and grid dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Call the kernel
    rgbToGrayscale << <gridDim, blockDim >> > (d_images, d_gray, width, height);

    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(error));
    }

    // Synchronize to make sure the kernel has finished
    hipDeviceSynchronize(); 
    return d_gray;
}


/*

__global__ void convert_to_float(unsigned char* d_images, float* d_images_float, unsigned char* d_label, float* d_label_float) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	d_images_float[idx] = (float)d_images[idx] / 255.0f; // Normalize to 0-1

	// convert labels to float
	d_label_float[idx] = (float)d_label[idx];
}

__global__ void convert_to_unsigened_char(float* d_images_float, unsigned char* d_images_new, 
								float* d_labels_float, unsigned char* d_labels_new) {

	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	d_images_new[idx] = (unsigned char)(d_images_float[idx]); // Normalize to 0-1

	// convert labels to float
	d_labels_new[idx] = (unsigned char)d_labels_float[idx];
}


void preprocess_images(unsigned char*& d_images, unsigned char*& d_labels) {
    // Step1. Convert to float
    float* d_images_float, * d_labels_float;
    hipMalloc(&d_images_float, IMG_SIZE * NUM_IMAGES * DATA_BATCHES * sizeof(float));
    hipMalloc(&d_labels_float, NUM_IMAGES * DATA_BATCHES * sizeof(float));

    int totalThreads = IMG_SIZE * NUM_IMAGES * DATA_BATCHES;
    int blockSize = 256;
    int gridSize = (totalThreads + blockSize - 1) / blockSize;

    convert_to_float << <gridSize, blockSize >> > (d_images, d_images_float, d_labels, d_labels_float);
    hipDeviceSynchronize();
    printf("Converted to float\n");

    // Step2. Convert back to unsigned char
    unsigned char* d_images_new, * d_labels_new;
    hipMalloc(&d_images_new, IMG_SIZE * NUM_IMAGES * DATA_BATCHES);
    hipMalloc(&d_labels_new, NUM_IMAGES * DATA_BATCHES);

    convert_to_unsigened_char << <gridSize, blockSize >> > (d_images_float, d_images_new, d_labels_float, d_labels_new);
    hipDeviceSynchronize();

    // Free old memory and update pointers
    hipFree(d_images);
    hipFree(d_labels);

    d_images = d_images_new;
    d_labels = d_labels_new;

    // Free intermediate float arrays
    hipFree(d_images_float);
    hipFree(d_labels_float);

    printf("Preprocessing done\n");
}

*/