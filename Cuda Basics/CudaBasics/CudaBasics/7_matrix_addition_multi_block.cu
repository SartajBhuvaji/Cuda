#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 4

__global__ void matAdd(int* A, int* B, int* C) {
    int i = threadIdx.x;
    int j = threadIdx.y;

    // Ensure that indices are within the matrix bounds
    if (i < N && j < N) {
        C[i * N + j] = A[i * N + j] + B[i * N + j];
    }
}

int main() {
    int size = N * N * sizeof(int);

    // Allocate host memory
    int h_A[N * N], h_B[N * N], h_C[N * N]; // Matrix A, B, C of size N x N

    // Initialize matrices on the host
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate device memory
    int* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(N, N); // We create a 2D grid of N x N threads
    int numBlocks = 1;

    // Launch the kernel
    matAdd << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C);

    // Copy result matrix from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Result matrix C:\n");
    for (int i = 0; i < N * N; ++i) {
        printf("%d ", h_C[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
