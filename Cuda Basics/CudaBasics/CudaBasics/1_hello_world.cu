#include "hip/hip_runtime.h"
﻿#include<hip/hip_runtime.h>
#include<stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void dkernel() {   // __global__ is a GPU kernel function specifier
	printf("Hello World");
}

int main() {
	dkernel << <1, 1 >> > ();  // <<<1,1>>> is a kernel launch configuration // 1 block and 1 thread
	hipDeviceSynchronize();   // hipDeviceSynchronize() is a function that waits for the device to finish its execution
	return 0;
}
