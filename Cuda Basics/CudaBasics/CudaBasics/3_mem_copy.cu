#include "hip/hip_runtime.h"
﻿#include<hip/hip_runtime.h>
#include<stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void mem_copy(int* a) {   // __global__ is a GPU kernel function specifier
	a[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {
	const int n = 10;
	int a[n], * d_a;

	hipMalloc(&d_a, n * sizeof(int));  // Allocate memory on the device, d_a is the pointer to the memory on the device
	mem_copy << <1, n >> > (d_a);

	hipDeviceSynchronize();
	hipMemcpy(a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from device to host
	hipFree(d_a); // Free the memory on the GPU

	for (int i = 0; i < n; i++) {
		printf("%d ", a[i]);
	}

	return 0;
}
