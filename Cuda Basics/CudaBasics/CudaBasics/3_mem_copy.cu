#include "hip/hip_runtime.h"
﻿#include<hip/hip_runtime.h>
#include<stdio.h>
#include "hip/hip_runtime.h"
#include ""

__global__ void mem_copy(int* a) {   // __global__ is a GPU kernel function specifier
	a[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main() {
	const int n = 10;
	int a[n], * d_a;

	hipMalloc(&d_a, n * sizeof(int));  // Allocate memory on the device, d_a is the pointer to the memory on the device
	mem_copy << <1, n >> > (d_a);

	hipDeviceSynchronize();
	hipMemcpy(a, d_a, n * sizeof(int), hipMemcpyDeviceToHost);  // Copy data from device to host
	hipFree(d_a); // Free the memory on the GPU

	for (int i = 0; i < n; i++) {
		printf("%d ", a[i]);
	}

	return 0;
}

// Typical CUDA program flow:
/*
1. Load data into host memory 
	- fread/ rand
2. Allocate memory on the device
	- cudaMemAlloc
3. Copy data from host to device memory
	- hipMemcpy
4. Execute the kernel
	- kernel<<<1, n>>>(d_a)
5. Copy the result back to the host
	- hipMemcpy
6. Free the device memory
	- hipFree
*/
