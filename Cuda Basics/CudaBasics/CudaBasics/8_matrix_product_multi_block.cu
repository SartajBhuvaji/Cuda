#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 32  // Size of matrix (N x N)

__global__ void matMul(int* A, int* B, int* C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate row index in multi-block grid
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Calculate column index in multi-block grid

    if (i < N && j < N) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = sum;
    }
}

int main() {
    int size = N * N * sizeof(int); // Size of matrices in bytes

    // Allocate host memory
    int h_A[N * N], h_B[N * N], h_C[N * N]; // Matrix A, B, C of size N x N but stored as 1D arrays 

    // Initialize matrices on the host
    for (int i = 0; i < N * N; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate device memory
    int* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy matrices from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Define block and grid dimensions
    dim3 threadsPerBlock(N, N);  // We create a 2D grid of N x N threads
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (N + threadsPerBlock.y - 1) / threadsPerBlock.y); // We calculate the number of blocks needed based on the matrix size

    matMul << <numBlocks, threadsPerBlock >> > (d_A, d_B, d_C);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print result matrix
    printf("Result matrix C:\n");
    for (int i = 0; i < N * N; ++i) {
        printf("%d ", h_C[i]);
        if ((i + 1) % N == 0) {
            printf("\n");
        }
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
