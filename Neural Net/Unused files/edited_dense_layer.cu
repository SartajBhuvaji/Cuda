#include <hip/hip_runtime.h>

#include <iostream>
#include <cmath>
#include <cstdlib>

// Kernel for dense layer forward pass
__global__ void denseForwardKernel(float* input, float* weights, float* biases, float* output, int inputSize, int outputSize, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < batchSize && col < outputSize) {
        float sum = 0.0f;
        for (int i = 0; i < inputSize; ++i) {
            sum += input[row * inputSize + i] * weights[i * outputSize + col];
        }
        output[row * outputSize + col] = sum + biases[col];
    }
}

// Kernel for calculating gradients of weights and biases
__global__ void denseBackwardKernel(float* d_input, float* d_gradients, float* d_grad_weights, float* d_grad_biases, int inputSize, int outputSize, int batchSize) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // batch index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // output neuron

    if (row < batchSize && col < outputSize) {
        // Compute gradient for biases
        atomicAdd(&d_grad_biases[col], d_gradients[row * outputSize + col]);

        // Compute gradient for weights
        for (int i = 0; i < inputSize; ++i) {
            float grad = d_input[row * inputSize + i] * d_gradients[row * outputSize + col];
            atomicAdd(&d_grad_weights[i * outputSize + col], grad);
        }
    }
}

class DenseLayer {
private:
    int inputSize, outputSize, batchSize;
    float* d_weights;       // Device memory for weights
    float* d_biases;        // Device memory for biases
    float* d_output;        // Device memory for output
    float* d_grad_weights;  // Device memory for weight gradients
    float* d_grad_biases;   // Device memory for bias gradients

public:
    DenseLayer(int inSize, int outSize, int batchSz)
        : inputSize(inSize), outputSize(outSize), batchSize(batchSz) {
        // Allocate memory for weights, biases, outputs, and gradients
        hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_biases, outputSize * sizeof(float));
        hipMalloc(&d_output, outputSize * batchSize * sizeof(float));
        hipMalloc(&d_grad_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_grad_biases, outputSize * sizeof(float));

        // Initialize weights and biases
        initializeParameters();
    }

    ~DenseLayer() {
        hipFree(d_weights);
        hipFree(d_biases);
        hipFree(d_output);
        hipFree(d_grad_weights);
        hipFree(d_grad_biases);
    }

    void initializeParameters() {
        // Initialize weights using Xavier initialization
        float scale = sqrt(2.0f / (inputSize + outputSize));
        float* h_weights = new float[inputSize * outputSize];
        float* h_biases = new float[outputSize];

        for (int i = 0; i < inputSize * outputSize; ++i) {
            h_weights[i] = scale * ((float)rand() / RAND_MAX * 2.0f - 1.0f);
        }
        for (int i = 0; i < outputSize; ++i) {
            h_biases[i] = 0.0f; // Initialize biases to zero
        }

        hipMemcpy(d_weights, h_weights, inputSize * outputSize * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_biases, h_biases, outputSize * sizeof(float), hipMemcpyHostToDevice);

        delete[] h_weights;
        delete[] h_biases;
    }

    float* forward(float* d_input) {
        // Perform matrix multiplication and add biases
        dim3 blockDim(16, 16);
        dim3 gridDim((outputSize + blockDim.x - 1) / blockDim.x, (batchSize + blockDim.y - 1) / blockDim.y);

        // Launch a kernel to perform the forward pass
        denseForwardKernel<<<gridDim, blockDim>>>(d_input, d_weights, d_biases, d_output, inputSize, outputSize, batchSize);

        hipDeviceSynchronize();
        return d_output;
    }

    void backward(float* d_input, float* d_gradients) {
        // Reset gradients to zero
        hipMemset(d_grad_weights, 0, inputSize * outputSize * sizeof(float));
        hipMemset(d_grad_biases, 0, outputSize * sizeof(float));

        dim3 blockDim(16, 16);
        dim3 gridDim((outputSize + blockDim.x - 1) / blockDim.x, (batchSize + blockDim.y - 1) / blockDim.y);

        // Launch a kernel to compute gradients
        denseBackwardKernel<<<gridDim, blockDim>>>(d_input, d_gradients, d_grad_weights, d_grad_biases, inputSize, outputSize, batchSize);

        hipDeviceSynchronize();
    }

    // Getters for gradients
    float* getGradWeights() const { return d_grad_weights; }
    float* getGradBiases() const { return d_grad_biases; }

    // Getters for weights and biases
    float* getWeights() const { return d_weights; }
    float* getBiases() const { return d_biases; }
}; 